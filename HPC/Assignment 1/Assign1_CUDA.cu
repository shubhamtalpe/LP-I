
#include <hip/hip_runtime.h>
#include<iostream>
#include<math.h>

#define n 1000

using namespace std;

//Calculate min element
__global__ void minimum(int *input){
    int threadId = threadIdx.x; //Thread Id
    int stepSize = 1;
    int numberOfThreads = blockDim.x;

    while(numberOfThreads > 0){
        if(threadId < numberOfThreads){
            int first = threadId*stepSize*2;
            int second = first + stepSize;
            if(input[second] < input[first]) input[first] = input[second];
        }
        stepSize <<= 1; //Multiply step size by 2
        numberOfThreads >>= 1; //Divide number of threads by 2
    }
}

//Calculate max element
__global__ void maximum(int *input){
    int threadId = threadIdx.x; //Thread Id
    int stepSize = 1;
    int numberOfThreads = blockDim.x;

    while(numberOfThreads > 0){
        if(threadId < numberOfThreads){
            int first = threadId*stepSize*2;
            int second = first + stepSize;
            if(input[second] > input[first]) input[first] = input[second];
        }
        stepSize <<= 1; //Multiply step size by 2
        numberOfThreads >>= 1; //Divide number of threads by 2
    }
}

//Calculate sum of all elements
__global__ void sum(int *input){
    int threadId = threadIdx.x; //Thread Id
    int stepSize = 1;
    int numberOfThreads = blockDim.x;

    while(numberOfThreads > 0){
        if(threadId < numberOfThreads){
            int first = threadId*stepSize*2;
            int second = first + stepSize;
            input[first] += input[second];
        }
        stepSize <<= 1; //Multiply step size by 2
        numberOfThreads >>= 1; //Divide number of threads by 2
    }
}

//Overload sum function for float
__global__ void sum(float *input){
    int threadId = threadIdx.x; //Thread Id
    int stepSize = 1;
    int numberOfThreads = blockDim.x;

    while(numberOfThreads > 0){
        if(threadId < numberOfThreads){
            int first = threadId*stepSize*2;
            int second = first + stepSize;
            input[first] += input[second];
        }
        stepSize <<= 1; //Multiply step size by 2
        numberOfThreads >>= 1; //Divide number of threads by 2
    }
}

__global__ void meanDiffSq(float *input, float mean){
    input[threadIdx.x] -= mean;
    input[threadIdx.x] *= input[threadIdx.x];
}

void copyIntToFloat(float *dest, int *src, int size){
    for(int i=0; i<size; i++){
        dest[i] = float(src[i]);
    }
}

//initialize array with random elements
void random_ints(int *input, int size){
    for(int i=0; i<size; i++){
        input[i] = rand()%100; // Generate random elements in range 0-99
    }
}

int main(){
    int size = n*sizeof(int);
    hipEvent_t start, stop;
    int *arr;
    int *arr_d, result;
    float time;

    arr = (int *)malloc(size); //Allocate memory on host

    random_ints(arr, n); //Initialize arr with random elements

    hipMalloc((void**)&arr_d, size); //Allocate memory on device

    //Calculate Minimum element
    hipMemcpy(arr_d, arr, size, hipMemcpyHostToDevice); //Copy elements from host to device
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    minimum<<<1, n/2>>>(arr_d); //Calculate min element
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipMemcpy(&result, arr_d, sizeof(int), hipMemcpyDeviceToHost); //Copy first element from device to host
    cout << "Minimum element is : " << result << endl;
    cout << "Time taken : " << time << " ms" << endl << endl;

    //Calculate Maximum element
    hipMemcpy(arr_d, arr, size, hipMemcpyHostToDevice); //Copy elements from host to device
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    maximum<<<1, n/2>>>(arr_d); //Calculate max element
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipMemcpy(&result, arr_d, sizeof(int), hipMemcpyDeviceToHost); //Copy first element from device to host
    cout << "Maximum element is : " << result << endl;
    cout << "Time taken : " << time << " ms" << endl << endl;

    //Calculate Sum of all elements
    hipMemcpy(arr_d, arr, size, hipMemcpyHostToDevice); //Copy elements from host to device
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    sum<<<1, n/2>>>(arr_d); //Calculate sum
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipMemcpy(&result, arr_d, sizeof(int), hipMemcpyDeviceToHost); //Copy first element from device to host
    cout << "Sum of all element is : " << result << endl;
    cout << "Time taken : " << time << " ms" << endl << endl;

    //Calculate mean
    float mean = float(result)/n;
    cout << "Mean of all element is : " << mean << endl << endl;

    //Calculate standard deviation
    float *arr_float;
    float *arr_std, stdValue;

    arr_float = (float*)malloc(n*sizeof(float));
    hipMalloc((void**)&arr_std, n*sizeof(float));
    copyIntToFloat(arr_float, arr, n); //Initialize float array
    
    hipMemcpy(arr_std, arr_float, n*sizeof(float), hipMemcpyHostToDevice); //Copy elements from host to device
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    meanDiffSq<<<1, n>>>(arr_std, mean); //Calculate mean difference
    sum<<<1, n/2>>>(arr_std); //Calculate sum
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipMemcpy(&stdValue, arr_std, sizeof(float), hipMemcpyDeviceToHost); //Copy first element from device to host
    stdValue /= n;
    stdValue = sqrt(stdValue);
    cout << "Standard deviation is : " << stdValue << endl;
    cout << "Time taken : " << time << " ms" << endl << endl;

    hipFree(arr_d);
    hipFree(arr_std);
    free(arr);
    free(arr_float);
}